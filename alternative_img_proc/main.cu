#include "hip/hip_runtime.h"
#include <time.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <cutil.h>
#include <ctime>
#include "bmplib.h"

unsigned int width, height;
int mask[3][3] = {1,2,1,
                  2,3,2,
                  1,2,1,};

int getPixel(unsigned char * arr, int col, int row){
    int sum = 0;
    for (int j=-1; j<=1; j++){
        for (int i=-1; i<=1; i++){
            int  color = arr[(row + j) * width + (col + i)];
            sum +=  color * mask[i+1][j+1];
        }
    }
return  sum /15;
}

void h_blur(unsigned char *arr, unsigned char *result){
    int  offset = 2 * width;
    for (int row=2; row < height -3; row++){
        for (int col=2; col <width -3; col++){
            result[offset + col] = getPixel(arr , col , row);
        }
        offset  += width;
    }
}
__global__ void d_blur(unsigned char *arr, unsigned char *result, int width, int height){
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    if (row < 2 || col < 2 || row  >= height  -3 || col  >= width  -3 )
        return;

    int  mask [3][3] = {1,2,1, 2,3,2, 1,2,1};

    int sum = 0;
    for (int j=-1; j<=1; j++){
        for (int i=-1; i<=1; i++){
            int  color = arr[(row + j) * width + (col + i)];
            sum +=  color * mask[i+1][j+1];
        }
    }
    result[row * width + col] = sum /15;
}

int main(int argc, char** argv){
    PIXEL* d_resultPixels;
    PIXEL* h_resultPixels;
    PIXEL* h_pixels = NULL;
    PIXEL* d_pixels = NULL;

    readFile("../example.bmp", &width, &height, &h_pixels);

    int img_size = sizeof(unsigned char *) * width * height;
    h_resultPixels = (unsigned char *)malloc(img_size);
    hipMalloc((void**)&d_pixels, img_size);
    hipMalloc((void**)&d_resultPixels, img_size);
    hipMemcpy(d_pixels, h_pixels, img_size, hipMemcpyHostToDevice);

    clock_t  starttime , endtime , difference;
    starttime = clock ();
    // apply  gaussian  blur
    h_blur(h_pixels , h_resultPixels);
    endtime = clock();
    difference = (endtime  - starttime);
    double  interval = difference / (double)CLOCKS_PER_SEC;
    printf("CPU  execution  time = %f ms\n", interval * 1000);

    writeFile("CPU_reslut.bmp", width, height, h_resultPixels);

    dim3  block (16 ,16);
    dim3  grid (width/16,  height /16);
    unsigned  int  timer = 0;
    cutCreateTimer (& timer);
    cutStartTimer(timer);
    /* CUDA  method  */
    d_blur  <<< grid , block  >>>(d_pixels , d_resultPixels , width , height);
    hipDeviceSynchronize ();
    cutStopTimer(timer);
    printf("CUDA  execution  time = %f ms\n",cutGetTimerValue(timer));
    hipMemcpy(h_resultPixels , d_resultPixels , ImageSize , hipMemcpyDeviceToHost);
    writeFile("GPU_reslut.bmp", width, height, h_resultPixels);

}
