#include "hip/hip_runtime.h"
/*
Team: Packers
PID: 5642858
Class: CDA4101   Section: U02
Affirmation:
"I affirm that this program is entirely
my own work and none of it is the work
of any other person."
*/

#include <stdlib.h>
#include <stdio.h>
#include <ctype.h>
#include <unistd.h>
#include "bmplib.h"

int img_pix[100000][100000] = {{0}};


//cuda function
__global__ void compressor(PIXEL * orig, int row, int col, int **img_pix){
    // int n = blockIdx.x * blockDim.x + threadIdx.x;
    // int k = blockIdx.y * blockDim.y + threadIdx.y;
    int rows, cols;
    int gpu_pix [3] = {};
    for(rows = 0; rows < row; rows++){
            for(cols = 0; cols < col; cols++){
                    PIXEL * test = orig + rows + cols;
                    gpu_pix[0] = (int)test -> r;
                    gpu_pix[1] = (int)test -> g;
                    gpu_pix[2] = (int)test -> b;
                    img_pix[blockIdx.x][threadIdx.x] = gpu_pix;
                    //img_pix[n][k][3] =;
                    printf("%d, %d, %d\n", img_pix[blockIdx.x][threadIdx.x][0], img_pix[blockIdx.x][threadIdx.x][1], img_pix[blockIdx.x][threadIdx.x][2]);
                    printf("rows: %d, cols: %d\n", rows, cols);

            }
        }

}
// middleware to handle gpu core and thread usage
void middleware(PIXEL* original, int rows, int cols, PIXEL* newImg){
    int numThreads = 1024;
    int numCores = (rows * cols) /  numThreads + 1;

    PIXEL* gpuAllocation;

    hipMalloc(&gpuAllocation, (rows * cols));
    hipMemcpy(gpuAllocation, original, (rows * cols), hipMemcpyHostToDevice);
    compressor<<<numCores, numThreads>>>(original, rows, cols, img_pix);
    hipMemcpy(newImg, gpuAllocation, (rows * cols), hipMemcpyDeviceToHost);
    hipFree(&gpuAllocation);
}
int main (int agrc, char **agrv){
    FILE *inputfile;
    inputfile = fopen("image_list.txt", "r");
    char image_name[256];
    fgets(image_name, 256, (FILE*)inputfile);
    while(!feof(inputfile)){
        int row, col;
        PIXEL *uncompressed, *compressed;
        readFile("example.bmp", &row, &col, &uncompressed);
        middleware(uncompressed, row, col, compressed);

        fgets(image_name, 256, (FILE*)inputfile);
    }
}
