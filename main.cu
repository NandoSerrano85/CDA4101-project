#include "hip/hip_runtime.h"
/*
Team: Packers
PID: 5642858
Class: CDA4101   Section: U02
Affirmation:
"I affirm that this program is entirely
my own work and none of it is the work
of any other person."
*/

#include <stdlib.h>
#include <stdio.h>
#include <ctype.h>
#include <unistd.h>
#include "bmplib.h"

#define TRUE = 1


//cuda function
__global__ void compressor(PIXEL * orig, int row, int col){
    int img_pix[10000][10000][3] = {{{0}}};
    int n = blockIdx.x * blockDim.x + threadIdx.x;
    int k = blockIdx.y * blockDim.y + threadIdx.y;
    int rows, cols;
    for(rows = 0; rows < row; rows++){
            for(cols = 0; cols < col; cols++){
                    PIXEL * test = orig + rows + cols;
                    img_pix[n][k][0] = (int)test -> r;
                    img_pix[n][k][1] = (int)test -> g;
                    img_pix[n][k][2] = (int)test -> b;
                    printf("%d, %d, %d\n", img_pix[n][k][0], img_pix[n][k][1], img_pix[n][k][2]);
                    printf("rows: %d, cols: %d\n", rows, cols);

            }
        }

}
// middleware to handle gpu core and thread usage
void middleware(PIXEL* original, int rows, int cols, PIXEL* newImg){
    int numThreads = 1024;
    int numCores = (rows * cols) /  numThreads + 1;

    PIXEL* gpuAllocation;

    hipMalloc(&gpuAllocation, (rows * cols));
    printf("test for hipMalloc\n");
    hipMemcpy(gpuAllocation, original, (rows * cols), hipMemcpyHostToDevice);
    printf("test for hipMemcpy to gpu\n");
    compressor<<<numCores, numThreads>>>(original, rows, cols);
    hipMemcpy(newImg, gpuAllocation, (rows * cols), hipMemcpyDeviceToHost);
    printf("test for hipMemcpy to cpu\n");
    hipFree(&gpuAllocation);
}
int main (int agrc, char **agrv){
    FILE *inputfile;
    inputfile = fopen("image_list.txt", "r");
    char image_name[256];
    fgets(image_name, 256, (FILE*)inputfile);
    while(!feof(inputfile)){
        int row, col;
        PIXEL *uncompressed, *compressed;
        readFile("example.bmp", &row, &col, &uncompressed);
        middleware(uncompressed, row, col, compressed);

        fgets(image_name, 256, (FILE*)inputfile);
    }
}
