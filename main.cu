#include "hip/hip_runtime.h"
/*
Team: Packers
PID: 5642858
Class: CDA4101   Section: U02
Affirmation:
"I affirm that this program is entirely
my own work and none of it is the work
of any other person."
*/

#include <stdlib.h>
#include <stdio.h>
#include <ctype.h>
#include <unistd.h>
#include "bmplib.h"

typedef struct {
  unsigned char r;
  unsigned char g;
  unsigned char b;
  unsigned int count;
} RESULT;

//cuda function
__global__ void compressor(PIXEL * orig, int width, int height, PIXEL *result){
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    int rgb[4];
    int n, k, count = 0, mask = 0;
    for(n = 0; n < 2; n++){
            for(k = 0; k < 2; k++){
                    PIXEL * test = orig + row + col;
                    rgb[0] = (int)test -> r;
                    rgb[1] = (int)test -> g;
                    rgb[2] = (int)test -> b;

                    test = orig + row + (col + 1);
                    if((int)test -> r >= (rgb[0]-4) && (int)test -> r <= (rgb[0]+4)){
                        count++;
                    }if((int)test -> r >= (rgb[1]-4) && (int)test -> r <= (rgb[1]+4)){
                        count++;
                    }if((int)test -> r >= (rgb[2]-4) && (int)test -> r <= (rgb[2]+4)){
                        count++;
                    }
                    if(count == 3){
                        mask++;
                        rgb[3] = mask;
                    }
                    printf("%d, %d, %d\n", rgb[0], rgb[1], rgb[2]);
                    printf("rows: %d, cols: %d\n", row, col);

            }
        }

}

// middleware to handle gpu core and thread usage
void middleware(PIXEL* original, int rows, int cols, PIXEL* result){
    dim3  block (16 ,16);
    dim3  grid (cols/16,  rows/16);
    int size = sizeof(unsigned char *) * rows * cols;
    // int numThreads = 1024;
    // int numCores = (rows * cols) /  numThreads + 1;

    PIXEL* gpu_picture;
    printf("middleware\n");
    hipMalloc((void **)&gpu_picture, size);
    hipMalloc((void **)&result, size);
    hipMemcpy(gpu_picture, original, size, hipMemcpyHostToDevice);
    compressor<<<grid, block>>>(gpu_picture, rows, cols, result);
    hipMemcpy(result, gpu_picture, (rows * cols), hipMemcpyDeviceToHost);
    hipFree(&gpu_picture);
}
int main (int agrc, char **agrv){
    FILE *inputfile;
    inputfile = fopen("image_list.txt", "r");
    char image_name[256];
    fgets(image_name, 256, (FILE*)inputfile);
    while(!feof(inputfile)){
        int row, col;
        PIXEL *uncompressed;
        PIXEL *compressed = NULL;
        readFile("example.bmp", &row, &col, &uncompressed);
        printf("main\n");
        middleware(uncompressed, row, col, compressed);

        fgets(image_name, 256, (FILE*)inputfile);
    }
}
