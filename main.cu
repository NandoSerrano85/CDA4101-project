#include "hip/hip_runtime.h"
/*
Team: Packers
PID: 5642858
Class: CDA4101   Section: U02
Affirmation:
"I affirm that this program is entirely
my own work and none of it is the work
of any other person."
*/

#include <stdlib.h>
#include <stdio.h>
#include <ctype.h>
#include <unistd.h>
#include "bmplib.h"

#define TRUE = 1


//cuda function
__global__ void compressor(PIXEL * orig, int row, int col){
    int img_pix[10000][10000][3] = {{{0}}};
    int n = blockIdx.x * blockDim.x + threadIdx.x;
    int k = blockIdx.y * blockDim.y + threadIdx.y;
    int rows, cols;
    for(rows = 0; rows < row; rows++){
            for(cols = 0; cols < col; cols++){
                    PIXEL * test = orig + rows + cols;
                    img_pix[n][k][0] = (int)test -> r;
                    img_pix[n][k][1] = (int)test -> g;
                    img_pix[n][k][2] = (int)test -> b;
                    printf("%d, %d, %d\n", img_pix[n][k][0], img_pix[n][k][1], img_pix[n][k][2]);
                    printf("rows: %d, cols: %d\n", rows, cols);

            }
        }

}
// middleware to handle gpu core and thread usage
void middleware(PIXEL* original, int rows, int cols, PIXEL* newImg){
    int numThreads = 512;
    int numCores = (rows * cols) /  numThreads + 1;

    int* gpuAllocation;

    hipMalloc(&gpuAllocation, (rows * cols));
    hipMemcpy(gpuAllocation, original, (rows * cols), hipMemcpyHostToDevice);
    compressor<<<numCores, numThreads>>>(original, rows, cols);
    hipMemcpy(newImg, gpuAllocation, (rows * cols), hipMemcpyDeviceToHost);
    hipFree(&gpuAllocation);
}
int main (int agrc, char **agrv){
    FILE *inputfile;
    inputfile = fopen("image_list.txt", "r");
    char image_name[256];
    fgets(image_name, 256, (FILE*)inputfile);
    while(!feof(inputfile)){
        int row, col;
        PIXEL *uncompressed, *compressed;
        readFile("example.bmp", &row, &col, &uncompressed);
        middleware(uncompressed, row, col, compressed);

        fgets(image_name, 256, (FILE*)inputfile);
    }
}
