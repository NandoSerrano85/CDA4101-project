#include "hip/hip_runtime.h"
/*
Team: Packers
PID: 5642858
Class: CDA4101   Section: U02
Affirmation:
"I affirm that this program is entirely
my own work and none of it is the work
of any other person."
*/

#include <stdlib.h>
#include <stdio.h>
#include <ctype.h>
#include <unistd.h>
#include "bmplib.h"

#define TRUE = 1


//cuda function
__global__ void compressor(PIXEL * orig, int row, int col){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int rows, cols;
    for(rows = 0; rows < row; rows++){
            for(cols = 0; cols < col; cols++){
                    PIXEL * test = orig + rows + cols;
                    printf("%d, %d, %d\n", test -> r, test -> g, test -> b);
                    printf("rows: %d, cols: %d\n", rows, cols);

            }
        }

}
// middleware to handle gpu core and thread usage
void middleware(PIXEL* original, int rows, int cols, PIXEL* new_image){
    int numThreads = 1024;
    int numCores = original*sizeof(int) /  numThreads + 1;

    int* gpuAllocation;

    hipMalloc(&gpuAllocation, original*sizeof(int));
    hipMemcpy(gpuAllocation, &original, original*sizeof(int), hipMemcpyHostToDevice);
    compressor<<<numCores, numThreads>>>(original, rows, cols);
    hipMemcpy(&new_image, gpuAllocation, original*sizeof(int), hipMemcpyDeviceToHost);
    hipFree(&gpuAllocation);
}
int main (int agrc, char **agrv){
    FILE *inputfile;
    inputfile = fopen("image_list.txt", "r");
    char image_name[256];
    fgets(image_name, 256, (FILE*)inputfile);
    while(!feof(inputfile)){
        int row, col;
        PIXEL *uncompressed, *compressed;
        readFile("example.bmp", &row, &col, &uncompressed);
        middleware(uncompressed, row, col, compressed);

        fgets(image_name, 256, (FILE*)inputfile);
    }
}
